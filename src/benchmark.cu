#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <string>
#include <iostream>

#include <skeletons.hpp>
#include <skeletons_gpu.hpp>
#include <marrow/timer.hpp>

using namespace std;


/////////////////
// The filter functions
/////////////////

template<typename... Ts>
using filter_func = bool (*) (Ts...);


template<typename T>
__device__  bool basic_op(T x, unsigned granularity) {

    int val = 0;
    for (unsigned i = 0; i < granularity; i++)
        val += i;

    return x > val;
}

template <typename T>
__device__ filter_func<T, unsigned> p_basic_op = basic_op<T>;

template<typename T>
__device__  bool sfu_op(T x, unsigned granularity) {

    int val = 0;
    for (unsigned i = 0; i < granularity; i++)
        val += sqrtf(i)/i;

    return x > val;
}

template <typename T>
__device__ filter_func<T, unsigned> p_sfu_op = sfu_op<T>;


/////////////////
// The main function
//   the benchmark receives four arguments
//      function: (0 - basic_op, 1 - special op)
//      nelems: Memory size - size of the container to process
//      granularity: Computational weight - number of iterations to be executed by each GPU thread.
//      nruns -- Number of times the benchmark is executed (optional, default is 1)
/////////////////

int main(int argc, char* argv[]) {
    if (argc < 4 || argc > 5) {
        printf("usage: %s function (0 - basic_op, 1 - special op) nelems granularity [nruns]\n", argv[0]);
        return 1;
    }

    unsigned char function = static_cast< unsigned char>(stoul (argv[1], nullptr,0));
    size_t nelems = stoul (argv[2], nullptr,0);
    unsigned granularity = stoul (argv[3], nullptr,0);
    unsigned nruns = argc == 3 ? 1 : stoul (argv[4], nullptr,0);

    auto in = make_shared<vector<float>>(nelems);

    marrow::timer<std::chrono::microseconds> t;

    if (function == 0) {
        for (unsigned i = 0; i < nruns; i++) {
            printf ("Run number %u\n", i);
            std::fill(in->begin(), in->end(), 1);

            t.start();
            auto out = cad::filter1_gpu(999.0f, p_basic_op<float>, *in, granularity);
            // cad::filter2_gpu(p_basic_op<float>, *in, granularity);
            t.stop();
        }
    }
    else {
        for (unsigned i = 0; i < nruns; i++) {
            printf ("Run number %u\n", i);
            std::fill(in->begin(), in->end(), 1);

            t.start();
            auto out = cad::filter1_gpu(999.0f, p_sfu_op<float>, *in, granularity);
            // cad::filter2_gpu(p_sfu_op<float>, *in, granularity);
            t.stop();
        }
    }

    t.average();
    t.std_deviation();
    t.output_stats(cout, marrow::main_stage, false);
}
